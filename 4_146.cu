#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
  
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library 
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
    nvcc -o 4_146 4_146.cu
    ./4_146
   
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

__device__ int is_a_match(char *attempt) {
	char plain_password1[] = "SU11";
	char plain_password2[] = "JA22";
	char plain_password3[] = "LL33";
	char plain_password4[] = "AL44";


	char *a = attempt;
	char *b = attempt;
	char *c = attempt;
	char *d = attempt;
	char *p1 = plain_password1;
	char *p2 = plain_password2;
	char *p3 = plain_password3;
	char *p4 = plain_password4;

	while(*a == *p1) { 
		if(*a == '\0') 
		{
			printf("Password: %s\n",plain_password1);
			break;
		}

		a++;
		p1++;
	}
	
	while(*b == *p2) { 
		if(*b == '\0') 
		{
			printf("Password: %s\n",plain_password2);
			break;
		}

		b++;
		p2++;
	}

	while(*c == *p3) { 
		if(*c == '\0') 
		{
			printf("Password: %s\n",plain_password3);
			break;
		}

		c++;
		p3++;
	}

	while(*d == *p4) { 
		if(*d == '\0') 
		{
			printf("Password: %s\n",plain_password4);
			return 1;
		}

		d++;
		p4++;
	}
	return 0;

}

__global__ void  kernel() {
	char i1,i2,i3,i4;

	char password[7];
	password[6] = '\0';

	int i = blockIdx.x+65;
	int j = threadIdx.x+65;
	char firstMatch = i; 
	char secondMatch = j; 

	password[0] = firstMatch;
	password[1] = secondMatch;
	for(i1='0'; i1<='9'; i1++){
		for(i2='0'; i2<='9'; i2++){
			for(i3='0'; i3<='9'; i3++){
				for(i4='0'; i4<='9'; i4++){
					password[2] = i1;
					password[3] = i2;
					password[4] = i3;
					password[5] = i4; 
					if(is_a_match(password)) {
					} 
					else {
	     			//printf("tried: %s\n", password);		  
					}
				}
			}
		}
	}
}

int time_difference(struct timespec *start, 
	struct timespec *finish, 
	long long int *difference) {
	long long int ds =  finish->tv_sec - start->tv_sec; 
	long long int dn =  finish->tv_nsec - start->tv_nsec; 
	if(dn < 0 ) {
		ds--;
		dn += 1000000000; 
	} 
	*difference = ds * 1000000000 + dn;
	return !(*difference > 0);
}


int main() {

	struct  timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);

	kernel <<<26,26>>>();
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

	return 0;
}


