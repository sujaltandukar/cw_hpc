#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
  
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library 
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
    nvcc -o 6_146 6_146.cu
    ./6_146
   
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

__device__ int is_a_match(char *attempt) {
	char plain_password1[] = "SU1234";
	char plain_password2[] = "JA2345";
	char plain_password3[] = "LL3456";
	char plain_password4[] = "AL4567";


	char *ab = attempt;
	char *bc = attempt;
	char *cd = attempt;
	char *de = attempt;
	char *ab1 = plain_password1;
	char *bc2 = plain_password2;
	char *cd3 = plain_password3;
	char *de4 = plain_password4;

	while(*ab == *ab1) { 
		if(*ab == '\0') 
		{
			printf("Password: %s\n",plain_password1);
			break;
		}

		ab++;
		ab1++;
	}
	
	while(*bc == *bc2) { 
		if(*bc == '\0') 
		{
			printf("Password: %s\n",plain_password2);
			break;
		}

		bc++;
		bc2++;
	}

	while(*cd == *cd3) { 
		if(*cd == '\0') 
		{
			printf("Password: %s\n",plain_password3);
			break;
		}

		cd++;
		cd3++;
	}

	while(*de == *de4) { 
		if(*de == '\0') 
		{
			printf("Password: %s\n",plain_password4);
			return 1;
		}

		de++;
		de4++;
	}
	return 0;

}

__global__ void  kernel() {
	char i1,i2,i3,i4;

	char password[7];
	password[6] = '\0';

	int i = blockIdx.x+65;
	int j = threadIdx.x+65;
	char firstMatch = i; 
	char secondMatch = j; 

	password[0] = firstMatch;
	password[1] = secondMatch;
	for(i1='0'; i1<='9'; i1++){
		for(i2='0'; i2<='9'; i2++){
			for(i3='0'; i3<='9'; i3++){
				for(i4='0'; i4<='9'; i4++){
					password[2] = i1;
					password[3] = i2;
					password[4] = i3;
					password[5] = i4; 
					if(is_a_match(password)) {
					} 
					else {
	     			//printf("tried: %s\n", password);		  
					}
				}
			}
		}
	}
}

// Calculate the difference between two times. Returns zero on
// success and the time difference through an argument. It will 
// be unsuccessful if the start time is after the end time.

int time_difference(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) {
  long long int d_sec =  finish->tv_sec - start->tv_sec; 
  long long int d_nsec =  finish->tv_nsec - start->tv_nsec; 

  if(d_nsec < 0 ) {
    d_sec--;
    d_nsec += 1000000000; 
  } 
  *difference = d_sec * 1000000000 + d_nsec;
  return !(*difference > 0);
}


int main() {

	struct  timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);

	kernel <<<26,26>>>();
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

	return 0;
}


